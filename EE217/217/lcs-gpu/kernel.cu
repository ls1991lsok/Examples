
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#define BLOCK_SIZE 16

__global__ void LCS_kenel(int map_row, int map_col, const char *stringA, const char *stringB, int *map, int i) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int cur_element = x + y * map_col;

    // if(x < map_col && y < map_row){
    //     if(map[cur_element] < 0){
    //         map[cur_element]++;
    //     }
    //     else if(map[cur_element] == 0){
    //         if(x == 0 || y == 0) map[cur_element] = 0;
    //         else if(stringA[x - 1] == stringB[y - 1]){
    //             int pre_element = (y - 1) * map_col + (x - 1);
    //             map[cur_element] = map[pre_element] + 1;
    //         }
    //         else{
    //             int cur_left = y * map_col + (x - 1);
    //             int cur_up = (y - 1) * map_col + x;
    //             if(map[cur_up] > map[cur_left]) map[cur_element] = map[cur_up];
    //             else map[cur_element] = map[cur_left];
    //         }
    //     }
    // }

    if( 0 < x && x < map_col && 0 < y && y < map_row && x + y == i){
        if(stringA[x - 1] == stringB[y - 1]){
            int pre_element = (y - 1) * map_col + (x - 1);
            map[cur_element] = map[pre_element] + 1;
        }else{
            int cur_left = y * map_col + (x - 1);
            int cur_up = (y - 1) * map_col + x;
            if(map[cur_up] > map[cur_left]) map[cur_element] = map[cur_up];
            else map[cur_element] = map[cur_left];
        }
    }
}

void computeLCS(int map_row, int map_col, const char *stringA, const char *stringB, int *map)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    dim3 DimGrid((map_col-1)/BLOCK_SIZE + 1, (map_row-1)/ BLOCK_SIZE + 1, 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Invoke CUDA kernel -----------------------------------------------------

    for(int i = 2; i < map_row + map_col; i++){
        LCS_kenel<<<DimGrid, DimBlock>>>(map_row, map_col, stringA, stringB, map, i);
    }
}

__global__ void LCS_kernel_optimized(int row, int col, const char *X, const char *Y, int *dp, int i) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = i - offset;
    int y = offset;
    int cur_element = x + y * col;

    if( 0 < x && x < col && 0 < y && y < row && x + y == i){
        if(X[x - 1] == Y[y - 1]){
            int pre_element = (y - 1) * col + (x - 1);
            dp[cur_element] = dp[pre_element] + 1;
        }else{
            int cur_left = y * col + (x - 1);
            int cur_up = (y - 1) * col + x;
            if(dp[cur_up] > dp[cur_left]) dp[cur_element] = dp[cur_up];
            else dp[cur_element] = dp[cur_left];
        }
    }
}

void computeLCS_optimized(int row, int col, const char *X, const char *Y, int *dp){
    const int block_size = 256;
    int max_n = sqrt((double)(row * row + col * col))+1;
    dim3 DimGrid((max_n-1)/block_size + 1, 1, 1);
    dim3 DimBlock(block_size, 1, 1);
    for(int i = 0; i < max_n; i++){
        LCS_kernel_optimized<<<DimGrid, DimBlock>>>(row, col, X, Y, dp, i);
    }
}

