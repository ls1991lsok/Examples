#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "kernel.cu"
#include "support.h"

int main (int argc, char *argv[])
{
    Timer timer;
    hipError_t cuda_ret;
    srand(1);

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    char *stringA_h, *stringB_h;
    char *stringA_d, *stringB_d;
    int *map_h, *map_d;
    size_t map_row, map_col, map_size;
    unsigned stringA_size, stringB_size;

    if (argc == 1) {
        stringA_size = stringB_size = 100;
    } else if (argc == 2) {
        stringA_size = stringB_size = atoi(argv[1]);
    } else if (argc == 3) {
        stringA_size = atoi(argv[1]);
        stringB_size = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./lcs-gpu                    # Both String Sizes are 10"
      "\n    Usage: ./lcs-gpu  <m>               # Both String Sizes are m"
      "\n    Usage: ./lcs-gpu  <m> <n>           # String A Size = m, String B Size = n"
      "\n");
        exit(0);
    }

    map_col = stringA_size + 1;
    map_row = stringB_size + 1;
    map_size = map_row * map_col;

    stringA_h = (char*)malloc(sizeof(char) * stringA_size);
    for(unsigned int i = 0; i < stringA_size; i++) stringA_h[i] = 'A' + rand() % 26;

    stringB_h = (char*)malloc(sizeof(char) * stringB_size);
    for(unsigned int i = 0; i < stringB_size; i++) stringB_h[i] = 'A' + rand() % 26;

    map_h = (int*)malloc(sizeof(int) * map_size);
    for(int i = 0; i < map_row; i++){
        for(int j = 0; j < map_col; j++){
            map_h[i * map_col + j] = 0;
        }
    }

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // printf("    String A :");
    // for(int i = 0; i < stringA_size; i++) printf("%c", stringA_h[i]);
    // printf("\n    String B :");
    // for(int i = 0; i < stringB_size; i++) printf("%c", stringB_h[i]);
    printf("\n    String A Length: %u\n    String B Length: %u\n", stringA_size, stringB_size);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&stringA_d, sizeof(char) * stringA_size);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&stringB_d, sizeof(char) * stringB_size);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&map_d, sizeof(int) * map_size);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(stringA_d, stringA_h, sizeof(char) * stringA_size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    cuda_ret = hipMemcpy(stringB_d, stringB_h, sizeof(char) * stringB_size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    cuda_ret = hipMemcpy(map_d, map_h, sizeof(int) * map_size, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    computeLCS(map_row, map_col, stringA_d, stringB_d, map_d);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(map_h, map_d, sizeof(int) * map_size, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the host");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);
    startTime(&timer);

    // verify(stringA_h, stringB_h, map_h, map_col, map_row);
    // printRes(map_h, map_col, map_row);                 //Uncomment this line to print out the map of LCS

    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("The Longest Common Subsequence Length is: %d\n", map_h[map_row * map_col - 1]);


    // Free memory ------------------------------------------------------------

    free(stringA_h);
    free(stringB_h);
    free(map_h);

    hipFree(stringA_d);
    hipFree(stringB_d);
    hipFree(map_d);

    return 0;

}