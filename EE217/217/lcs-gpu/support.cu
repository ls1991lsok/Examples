#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include "support.h"

void verify(const char *stringA, const char *stringB, int *map, unsigned int map_col, unsigned int map_row){

  const float relativeTolerance = 1e-6;
  unsigned int count = 0;
  int *dp = (int*)malloc(map_row * map_col * sizeof(int));
  int *print = (int*)malloc(map_row * map_col * sizeof(int));

  for(int i = 0; i < map_row; i++){
    for(int j = 0; j < map_col; j++){
      dp[i * map_col + j] = 0;
      print[i * map_col + j] = 0;
    }
  }

  for(int i = 1; i < map_row; i++) {
    for(int j = 1; j < map_col; j++) {
      if(stringB[i - 1] == stringA[j - 1]){
        dp[i * map_col + j] = 1 + dp[(i - 1) * map_col + (j - 1)];
        print[i * map_col + j] = 5;
      }
      else{
        dp[i * map_col + j] = max(dp[(i - 1) * map_col + j], dp[i * map_col + (j - 1)]);
        print[i * map_col + j] = dp[(i - 1) * map_col + j] >= dp[i * map_col + (j - 1)]? 7 : 9;
      }
      count++;
      float relativeError = dp[i * map_col + j] - map[i*(map_col) + j];
      if (relativeError > relativeTolerance
        || relativeError < -relativeTolerance) {
        printf("\nTEST FAILED %u\n\n",count);
        exit(1);
      }
    }
  }
  printf("TEST PASSED %u\n\n", count);
  printf("The Longest Common Subsequence is:  ");
  printLCS((int*)print, stringB, map_col, map_row, map_col);
  printf("\n");

  free(dp);
  free(print);
}

void printLCS(int *print, const char *stringB, int row_size, int map_row, int map_col){
  int stringB_size = map_row - 1;
  int stringA_size = map_col - 1;
  if(stringB_size == 0 || stringA_size == 0) return;
  if(print[stringB_size * row_size + stringA_size] == 5){
    printLCS(print, stringB, row_size, map_row - 1, map_col - 1);
    printf("%c", stringB[stringB_size - 1]);
  }else if(print[stringB_size * row_size + stringA_size] == 7){
    printLCS(print, stringB, row_size, map_row - 1, map_col);
  }else if(print[stringB_size * row_size + stringA_size] == 9){
    printLCS(print, stringB, row_size, map_row, map_col  - 1);
  }
}

void printRes(int *map, unsigned int map_col, unsigned int map_row){
    printf("\n");
    for(int i = 0; i < map_row; i++){
        printf("[");
        for(int j = 0; j < map_col; j++){
            printf(" %d ", map[i*map_col + j]);
        }
        printf("]\n");
    }
}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

